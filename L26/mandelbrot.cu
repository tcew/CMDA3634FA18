/* 

To compile:

   nvcc --arch=sm_60 -O3 -o mandelbrot mandelbrot.c -lm

To create an image with 4096 x 4096 pixels 

    ./mandelbrot 4096 4096 

*/


#include <hip/hip_runtime.h>
#include <math.h>
#include <stdio.h>
#include <stdlib.h>
#include <time.h>

int writeMandelbrot(const char *fileName, int width, int height, float *img, int minI, int maxI);

// Q2a: add include for CUDA header file here:

#define MXITER 1000

typedef struct {
  
  double r;
  double i;
  
}complex_t;

// return iterations before z leaves mandelbrot set for given c
__device__ int testpoint(complex_t c){

  int iter;
  complex_t z;
  double temp;
  
  z = c;
  
  for(iter=0; iter<MXITER; iter++){  
    temp = (z.r*z.r) - (z.i*z.i) + c.r;
    
    z.i = z.r*z.i*2. + c.i;
    z.r = temp;
    
    if((z.r*z.r+z.i*z.i)>4.0){
      return iter;
    }
  }
  return iter; 
}

// perform Mandelbrot iteration on a grid of numbers in the complex plane
// record the  iteration counts in the count array
__global__ void mandelbrotKernel(int Nre, int Nim, complex_t cmin, complex_t dc, float *count){ 

  // Q2c: replace this loop with a CUDA kernel
  //  for(int n=0;n<Nim;++n){
  //    for(int m=0;m<Nre;++m){

  int n = threadIdx.y + blockIdx.y*blockDim.y;
  int m = threadIdx.x + blockIdx.x*blockDim.x;
  if(n<Nim && m<Nre){
    complex_t c;
    
    c.r = cmin.r + dc.r*m;
    c.i = cmin.i + dc.i*n;
    
    count[m+n*Nre] = (float) testpoint(c);
  }
}


int main(int argc, char **argv){

  // to create a 4096x4096 pixel image 
  // usage: ./mandelbrot 4096 4096 

  int Nre = (argc==3) ? atoi(argv[1]): 4096;
  int Nim = (argc==3) ? atoi(argv[2]): 4096;

  // Q2b: set the number of threads per block and the number of blocks here:
  
  // storage for the iteration counts
  float *count;
  count = (float*) malloc(Nre*Nim*sizeof(float));  

  // Parameters for a bounding box for "c" that generates an interesting image
  const float centRe = -.759856, centIm= .125547;
  const float diam  = 0.151579;

  complex_t cmin; 
  complex_t cmax;
  complex_t dc;

  cmin.r = centRe - 0.5*diam;
  cmax.r = centRe + 0.5*diam;
  cmin.i = centIm - 0.5*diam;
  cmax.i = centIm + 0.5*diam;

  //set step sizes
  dc.r = (cmax.r-cmin.r)/(Nre-1);
  dc.i = (cmax.i-cmin.i)/(Nim-1);

  hipEvent_t start, end;

  hipEventCreate(&start);
  hipEventCreate(&end);

  float *c_count;
  hipMalloc(&c_count, Nre*Nim*sizeof(float));

  int BX = 16, BY = 16;
  int GX = (Nre+BX-1)/BX, GY = (Nim+BY-1)/BY;

  dim3 B(BX,BY,1);
  dim3 G(GX,GY,1);

  // mark event 
  hipEventRecord(start);

  // compute mandelbrot set
  mandelbrotKernel <<< G, B >>> (Nre, Nim, cmin, dc, c_count); 
  
  // mark event 
  hipEventRecord(end);

  // copy from the GPU back to the host here
  hipMemcpy(count, c_count, Nre*Nim*sizeof(float), hipMemcpyDeviceToHost);

  // print elapsed time
  float elapsed;
  hipEventSynchronize(end);
  hipEventElapsedTime(&elapsed, start, end);
  elapsed /= 1000.;
  printf("elapsed = %f\n", elapsed);

  // output mandelbrot to ppm format image
  printf("Printing mandelbrot.ppm...");
  writeMandelbrot("mandelbrot.ppm", Nre, Nim, count, 0, 80);
  printf("done.\n");

  free(count);

  exit(0);
  return 0;
}  


/* Output data as PPM file */
void saveppm(const char *filename, unsigned char *img, int width, int height){

  /* FILE pointer */
  FILE *f;
  
  /* Open file for writing */
  f = fopen(filename, "wb");
  
  /* PPM header info, including the size of the image */
  fprintf(f, "P6 %d %d %d\n", width, height, 255);

  /* Write the image data to the file - remember 3 byte per pixel */
  fwrite(img, 3, width*height, f);

  /* Make sure you close the file */
  fclose(f);
}



int writeMandelbrot(const char *fileName, int width, int height, float *img, int minI, int maxI){

  int n, m;

  unsigned char *rgb   = (unsigned char*) calloc(3*width*height, sizeof(unsigned char));
  
  for(n=0;n<height;++n){
    for(m=0;m<width;++m){
      int id = m+n*width;

      int I = (int) (768*sqrt((double)(img[id]-minI)/(maxI-minI)));
      
      // change this to change palette
      if(I<256)      rgb[3*id+2] = 255-I;
      else if(I<512) rgb[3*id+1] = 511-I;
      else if(I<768) rgb[3*id+0] = 767-I;
      else if(I<1024) rgb[3*id+0] = 1023-I;
      else if(I<1536) rgb[3*id+1] = 1535-I;
      else if(I<2048) rgb[3*id+2] = 2047-I;

    }
  }

  saveppm(fileName, rgb, width, height);

  free(rgb);
  
  return 1;
}


