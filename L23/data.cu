
#include <stdlib.h>
#include <stdio.h>

#include <hip/hip_runtime.h>

__global__ void vectorAddKernel(int N, int *c_a, int *c_b, int *c_c){
  
  int threadIndex = threadIdx.x;
  int blockIndex  = blockIdx.x;
  int threadCount = blockDim.x;

  int n = threadIndex + threadCount*blockIndex;
  
  // check if n is in [0,N)
  if(n<N)
    c_c[n] = c_a[n] + c_b[n];


}


int main(int argc, char **argv){

  int N = 4097;

  int threadsPerBlock = 32;
  int blocks = (N+threadsPerBlock-1)/threadsPerBlock;
  
  // ON HOST
  int *h_a = (int*) malloc(N*sizeof(int));
  int *h_b = (int*) malloc(N*sizeof(int));
  int *h_c = (int*) malloc(N*sizeof(int));
  
  int n;
  for(n=0;n<N;++n){
    h_a[n] = 1 + n;
    h_b[n] = 1 - n;
  }

  // ON DEVICE
  int *c_a, *c_b, *c_c;

  hipMalloc(&c_a, N*sizeof(int));
  hipMalloc(&c_b, N*sizeof(int));
  hipMalloc(&c_c, N*sizeof(int));
  
  hipMemcpy(c_a, h_a, N*sizeof(int), hipMemcpyHostToDevice);
  hipMemcpy(c_b, h_b, N*sizeof(int), hipMemcpyHostToDevice);

  // INITIATE KERNEL ON DEVICE
  vectorAddKernel <<< blocks, threadsPerBlock >>> (N, c_a, c_b, c_c);

  // COPY DATA FROM DEVICE TO HOST
  hipMemcpy(h_c, c_c, N*sizeof(int), hipMemcpyDeviceToHost);

  // PRINT ENTRIES
  for(n=0;n<5;++n){
    printf("c[%d] = %d\n", n, h_c[n]);
  }

  hipDeviceSynchronize();
  hipFree(c_a);
  hipFree(c_b);
  hipFree(c_c);
}
